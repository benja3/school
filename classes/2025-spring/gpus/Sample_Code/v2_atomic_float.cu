
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DIM 512

struct Lock {
  int *mutex;
  Lock( void ) {
    int state = 0;
    hipMalloc( (void**)& mutex, sizeof(int) );
    hipMemcpy( mutex, &state, sizeof(int), hipMemcpyHostToDevice );
  }

  ~Lock( void ) {
    hipFree( mutex );
  }

  __device__ void lock( void ) {
    while( atomicCAS( mutex, 0, 1 ) != 0 );
  }

  __device__ void unlock( void ) {
   atomicExch( mutex, 0 );
  }
};

// Forward function declarations
float CPU_big_dot(float *A, float *B, int N);
float *get_random_vector(int N);
void die(const char *message);

template <unsigned int iBlockSize>
__global__ void kernel1(float *A, float *B, float *C, unsigned int  N)
{
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = A + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < N)
    {
        float a1 = A[idx] * B[idx];
        float a2 = A[idx + blockDim.x] * B[idx + blockDim.x];
        float a3 = A[idx + 2 * blockDim.x] * B[idx + 2 * blockDim.x];
        float a4 = A[idx + 3 * blockDim.x] * B[idx + 3 * blockDim.x];
        float b1 = A[idx + 4 * blockDim.x] * B[idx + 4 * blockDim.x];
        float b2 = A[idx + 5 * blockDim.x] * B[idx + 5 * blockDim.x];
        float b3 = A[idx + 6 * blockDim.x] * B[idx + 6 * blockDim.x];
        float b4 = A[idx + 7 * blockDim.x] * B[idx + 7 * blockDim.x];
        A[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (iBlockSize >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (iBlockSize >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (iBlockSize >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) C[blockIdx.x] = smem[0];
}

template <unsigned int iBlockSize>
__global__ void kernel2(float *A, float *B, float *sum, unsigned int  N)
{
    __shared__ float smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = A + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < N)
    {
        float a1 = A[idx] * B[idx];
        float a2 = A[idx + blockDim.x] * B[idx + blockDim.x];
        float a3 = A[idx + 2 * blockDim.x] * B[idx + 2 * blockDim.x];
        float a4 = A[idx + 3 * blockDim.x] * B[idx + 3 * blockDim.x];
        float b1 = A[idx + 4 * blockDim.x] * B[idx + 4 * blockDim.x];
        float b2 = A[idx + 5 * blockDim.x] * B[idx + 5 * blockDim.x];
        float b3 = A[idx + 6 * blockDim.x] * B[idx + 6 * blockDim.x];
        float b4 = A[idx + 7 * blockDim.x] * B[idx + 7 * blockDim.x];
        A[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (iBlockSize >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (iBlockSize >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (iBlockSize >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) atomicAdd(sum, smem[0]);
}

int main(int argc, char **argv) {
	// Seed the random generator (use a constant here for repeatable results)
	srand(10);

	// Determine the vector length
	int N = 1 << 24;  // default value

	// Generate two random vectors
	float *A_CPU = get_random_vector(N);
	float *B_CPU = get_random_vector(N);
	
	// Compute their dot product on the CPU
	float sumCPU = CPU_big_dot(A_CPU, B_CPU, N);
        printf("dot product with CPU= %f\n", sumCPU);	

        // execution configuration
        int blocksize = DIM;   // initial block size

        dim3 block (blocksize, 1);
        dim3 grid  ((N + block.x - 1) / block.x, 1);
        printf("grid %d block %d\n", grid.x, block.x);

	// Allocate GPU memory for the inputs and the result
	int vector_size = N * sizeof(float);
	float *A_GPU, *B_GPU, *C_GPU;
	if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &B_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &C_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
 
	// Transfer the input vectors to GPU memory
	hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        hipEvent_t start, stop;
        float kernel1_time;
        hipEventCreate( &start);
        hipEventCreate( &stop);
        hipEventRecord( start, 0);
        // Execute the kernel to compute the vector dot product on the GPU
        switch (blocksize) {
          case 1024:
            kernel1<1024><<<grid.x/8, block>>>(A_GPU, B_GPU, C_GPU, N);
            break;
          case 512:
            kernel1<512><<<grid.x/8, block>>>(A_GPU, B_GPU, C_GPU, N);
            break;
          case 256:
            kernel1<256><<<grid.x/8, block>>>(A_GPU, B_GPU, C_GPU, N);
            break;
          case 128:
            kernel1<128><<<grid.x/8, block>>>(A_GPU, B_GPU, C_GPU, N);
            break;
          case 64:
            kernel1<64><<<grid.x/8, block>>>(A_GPU, B_GPU, C_GPU, N);
            break;
        }
        hipDeviceSynchronize();
        hipEventRecord( stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime( &kernel1_time, start, stop);
        printf("Time takes for kernel1: %f ms\n", kernel1_time);

	// Check for kernel errors
	hipError_t error = hipGetLastError();
	if (error) {
	  char message[256];
	  sprintf(message, "CUDA error: %s", hipGetErrorString(error));
	  die(message);
	}
	
	// Allocate CPU memory for the result
	float *C_CPU = (float *) malloc(vector_size);
	if (C_CPU == NULL) die("Error allocating CPU memory");
	
	// Transfer the result from the GPU to the CPU
	hipMemcpy(C_CPU, C_GPU, vector_size, hipMemcpyDeviceToHost);
	
  // Do the summation of multiplication in CPU
  float sumKernel1 = 0;
	for (int i = 0; i < grid.x / 8; i++) sumKernel1 += C_CPU[i]; 
        printf("dot product with kernel1 = %f\n", sumKernel1);	

  float sumKernel2 = 0;
  float *sumKernel2_GPU;
	if (hipMalloc((void **) &sumKernel2_GPU, sizeof(float)) != hipSuccess) die("Error allocating GPU memory");

	// Transfer the input vectors to GPU memory
	hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
	hipMemcpy(sumKernel2_GPU, &sumKernel2, sizeof(float), hipMemcpyHostToDevice);

  float kernel2_time;
  hipEventRecord( start, 0);
  // Execute the kernel to compute the vector dot product on the GPU
  switch (blocksize) {
    case 1024:
      kernel2<1024><<<grid.x/8, block>>>(A_GPU, B_GPU, sumKernel2_GPU, N);
      break;
    case 512:
      kernel2<512><<<grid.x/8, block>>>(A_GPU, B_GPU, sumKernel2_GPU, N);
      break;
    case 256:
      kernel2<256><<<grid.x/8, block>>>(A_GPU, B_GPU, sumKernel2_GPU, N);
      break;
    case 128:
      kernel2<128><<<grid.x/8, block>>>(A_GPU, B_GPU, sumKernel2_GPU, N);
      break;
    case 64:
      kernel2<64><<<grid.x/8, block>>>(A_GPU, B_GPU, sumKernel2_GPU, N);
      break;
  }
  hipDeviceSynchronize();
  hipEventRecord( stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime( &kernel2_time, start, stop);
  printf("Time takes for kernel2: %f ms\n", kernel2_time);

  // Check for kernel errors
  hipGetLastError();
  if (error) {
    char message[256];
    sprintf(message, "CUDA error: %s", hipGetErrorString(error));
    die(message);
  }

  // Transfer the result from teh GPU to the CPU
  hipMemcpy(&sumKernel2, sumKernel2_GPU, sizeof(float), hipMemcpyDeviceToHost);
  printf("dot product with kernel2 = %f\n", sumKernel2);	

	// Compute the speedup or slowdown
	if (kernel1_time > kernel2_time) printf("kernel1 outperformed kernel2 by %.2fx\n", (float) kernel2_time / (float) kernel1_time);
	else printf("kernel2 outperformed kernel1 by %.2fx\n", (float) kernel1_time / (float) kernel2_time);
	
        hipFree(A_GPU);
        hipFree(B_GPU);
        hipFree(C_GPU);
}

// Returns the vector dot product of A and B
float CPU_big_dot(float *A, float *B, int N) {	
	// Compute the dot product
        float sum = 0;
	for (int i = 0; i < N; i++) sum += A[i] * B[i];
	
	// Return the result
	return sum;
}

// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
	if (N < 1) die("Number of elements must be greater than zero");
	
	// Allocate memory for the vector
	float *V = (float *) malloc(N * sizeof(float));
	if (V == NULL) die("Error allocating CPU memory");
	
	// Populate the vector with random numbers
	for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
	
	// Return the randomized vector
	return V;
}

// Prints the specified message and quits
void die(const char *message) {
	printf("%s\n", message);
	exit(1);
}

